// CUDA BASIC CODE

// DEFINE HEADER FILE IN APPLICATION
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <getopt.h>
#include <unistd.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>

// CUDA PARALLEL LIB
#include <hip/hip_runtime.h>

#include <stdio.h>

// DEFINE COLOR CODE
#define RED   "\x1B[31m"
#define GRN   "\x1B[32m"
#define YEL   "\x1B[33m"
#define BLU   "\x1B[34m"
#define MAG   "\x1B[35m"
#define CYN   "\x1B[36m"
#define WHT   "\x1B[37m"
#define RESET "\x1B[0m"

__global__ void gpuCompute()
{
printf (GRN"[ GPU ] Executing Statement ...! \n"RESET);
}

int main()
{
        struct timeval  tv1, tv2;
        gettimeofday(&tv1, NULL);
        
        gpuCompute<<<2,4>>>();
        hipDeviceSynchronize();
        hipDeviceReset();

        gettimeofday(&tv2, NULL);
        printf (CYN"GPU EXECUTION TIME : %f (Microseconds)\n",
        (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 +
        (double) (tv2.tv_sec - tv1.tv_sec));

return 0;
}
